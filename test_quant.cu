// #include <cuda_fp16.h>
#include <hip/hip_fp4.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdint.h>
#include <vector>
#include <random> 

__global__ void compute_max_val(const half* x, float* max_val, int group_size, int n_groups) {
    int group_id = blockIdx.x;
    int tid = threadIdx.x;
    int start_idx = group_id * group_size;
    int idx = start_idx + tid;
    // printf("hello");
    extern __shared__ float shared_max[];
    float local_max = 0.0f;

    if (idx < (group_id + 1) * group_size) {
        local_max = fabsf(__half2float(x[idx]));
    }
    // printf("hello");
    shared_max[tid] = local_max;
    __syncthreads();

    // 归约求最大值
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (shared_max[tid + s] > shared_max[tid]) {
                shared_max[tid] = shared_max[tid + s];
            }
        }
        __syncthreads();
    }
    if (tid == 0) {
        max_val[group_id] = fmaxf(shared_max[0], 1e-5f);
    }
}

__device__ __hip_fp4_e2m1 lookup_quantize(float x_scaled) {
    const float sorted_candidates[] = {0.0f, 0.5f, 1.0f, 1.5f, 2.0f, 3.0f, 4.0f, 6.0f};
    float abs_x = fabsf(x_scaled);
    float closest = sorted_candidates[0];
    float min_diff = fabsf(abs_x - closest);

    for (int i = 1; i < 8; ++i) {
        float diff = fabsf(abs_x - sorted_candidates[i]);
        if (diff < min_diff) {
            min_diff = diff;
            closest = sorted_candidates[i];
        }
    }
    if (abs_x > sorted_candidates[7]) closest = sorted_candidates[7];
    // closest = 0;
    return __hip_fp4_e2m1(__float2half_rn(closest * copysignf(1.0f, x_scaled)));
}

// 比较查表法与 Intrinsic 法的比特差异
__global__ void compare_fp4_bits(const __half* input, 
                                uint8_t* lookup_result, 
                                uint8_t* intrinsic_result, 
                                int group_size, 
                                const float* max_vals) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("Thread %d: start\n", idx);

    int group_id = idx / group_size;
    // printf("hello");
    // printf("group_id %d", group_id);
    // printf("gridDim.x : %d", gridDim.x);
    // if (group_id >= gridDim.x) return;

    // 计算缩放因子
    float max_val = max_vals[group_id];
    float scale = max_val / 6.0f;

    // 缩放输入值
    float x_scaled = __half2float(input[idx]) / scale;

    // 查表法量化
    __hip_fp4_e2m1 q_lookup = lookup_quantize(x_scaled);
    // Intrinsic 法直接转换
    __hip_fp4_e2m1 q_intrinsic = __hip_fp4_e2m1(__float2half_rn(x_scaled));

    // 将 FP4 值按比特存储到结果中
    int byte_idx = idx / 2;
    int shift = (idx % 2) * 4;
    // printf("hello %d ", idx);
    lookup_result[byte_idx] |= (static_cast<uint8_t>(q_lookup) & 0xF) << shift;
    intrinsic_result[byte_idx] |= (static_cast<uint8_t>(q_intrinsic) & 0xF) << shift;
}


void check_bit_errors(const uint8_t* d_lookup, 
                     const uint8_t* d_intrinsic, 
                     int total_elements) {
    std::vector<uint8_t> h_lookup(total_elements / 2);
    std::vector<uint8_t> h_intrinsic(total_elements / 2);
    hipMemcpy(h_lookup.data(), d_lookup, total_elements / 2, hipMemcpyDeviceToHost);
    hipMemcpy(h_intrinsic.data(), d_intrinsic, total_elements / 2, hipMemcpyDeviceToHost);

    // 打印 h_lookup 和 h_intrinsic 的内容
    printf("\n=== h_lookup (查表法) ===\n");
    for (size_t i = 0; i < h_lookup.size(); ++i) {
        printf("%02X ", h_lookup[i]);
        if ((i + 1) % 8 == 0) printf("\n");  // 每8个字节换行
    }

    printf("\n\n=== h_intrinsic (Intrinsic法) ===\n");
    for (size_t i = 0; i < h_intrinsic.size(); ++i) {
        printf("%02X ", h_intrinsic[i]);
        if ((i + 1) % 8 == 0) printf("\n");
    }
    printf("\n");

    int errors = 0;
    for (size_t i = 0; i < h_lookup.size(); ++i) {
        if (h_lookup[i] != h_intrinsic[i]) {
            // 检查每个 4-bit 单元
            uint8_t lookup_low = h_lookup[i] & 0xF;
            uint8_t intrinsic_low = h_intrinsic[i] & 0xF;
            uint8_t lookup_high = (h_lookup[i] >> 4) & 0xF;
            uint8_t intrinsic_high = (h_intrinsic[i] >> 4) & 0xF;

            if (lookup_low != intrinsic_low) errors++;
            if (lookup_high != intrinsic_high) errors++;
        }
    }
    printf("Bit-level errors: %d/%d (%.2f%%)\n", 
           errors, total_elements, 100.0f * errors / total_elements);
}


int main() {
    const int group_size = 16;
    const int n_groups = 16;
    const int total_elements = group_size * n_groups;

    // 初始化输入数据（正态分布）
    std::random_device rd;
    std::mt19937 gen(rd());
    
    const float mu = 0.0f;      // 均值
    const float sigma = 1.666f; // 标准差

    std::normal_distribution<float> dist(mu, sigma);
    std::vector<__half> h_input(total_elements);

    for (auto& val : h_input) {
        float num = dist(gen);
        val = __float2half_rn(num);
        // val = 100;
    }

    __half* d_input;
    float* d_max_vals;
    uint8_t* d_lookup_result, *d_intrinsic_result;
    hipMalloc(&d_input, total_elements * sizeof(__half));
    hipMalloc(&d_max_vals, n_groups * sizeof(float));
    hipMalloc(&d_lookup_result, total_elements / 2);
    hipMalloc(&d_intrinsic_result, total_elements / 2);
    
    hipMemset(d_lookup_result, 0, total_elements / 2);
    hipMemset(d_intrinsic_result, 0, total_elements / 2);

    hipMemcpy(d_input, h_input.data(), total_elements * sizeof(__half), hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((total_elements + block.x - 1) / block.x);

    // 计算group-wise的最大值
    compute_max_val<<<n_groups, group_size, group_size * sizeof(float)>>>(d_input, d_max_vals, group_size, n_groups);
    hipDeviceSynchronize();
    hipError_t err = hipPeekAtLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
    // printf("hello");
    //分别计算查表法和intrinsic法的结果
    compare_fp4_bits<<<grid, block>>>(d_input, d_lookup_result, d_intrinsic_result, group_size, d_max_vals);
    hipDeviceSynchronize();
    err = hipPeekAtLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
    //比较两者的bit-level error
    check_bit_errors(d_lookup_result, d_intrinsic_result, total_elements);
    // Bit-level errors: 0/256 (0.00%)
    hipFree(d_input);
    hipFree(d_max_vals);
    hipFree(d_lookup_result);
    hipFree(d_intrinsic_result);

    return 0;
}

// === h_lookup (查表法) ===
// 00 00 03 04 03 03 00 06 
// 06 00 06 00 00 00 00 00 
// 03 00 01 04 02 00 00 02 
// 02 01 03 00 00 04 04 00 
// 03 00 00 01 00 00 00 04 
// 00 04 00 00 01 00 00 00 
// 03 00 03 00 00 00 00 00 
// 00 00 02 00 00 00 04 00 
// 00 00 00 00 00 00 03 00 
// 00 00 00 04 01 06 00 01 
// 02 01 00 00 06 00 00 00 
// 00 02 00 00 01 00 00 00 
// 03 00 00 00 00 00 00 02 
// 00 04 04 00 06 00 00 00 
// 02 00 01 00 00 03 06 00 
// 00 00 00 06 06 02 01 00 


// === h_intrinsic (Intrinsic法) ===
// 00 00 03 04 03 03 00 06 
// 06 00 06 00 00 00 00 00 
// 03 00 01 04 02 00 00 02 
// 02 01 03 00 00 04 04 00 
// 03 00 00 01 00 00 00 04 
// 00 04 00 00 01 00 00 00 
// 03 00 03 00 00 00 00 00 
// 00 00 02 00 00 00 04 00 
// 00 00 00 00 00 00 03 00 
// 00 00 00 04 01 06 00 01 
// 02 01 00 00 06 00 00 00 
// 00 02 00 00 01 00 00 00 
// 03 00 00 00 00 00 00 02 
// 00 04 04 00 06 00 00 00 
// 02 00 01 00 00 03 06 00 
// 00 00 00 06 06 02 01 00 

// Bit-level errors: 0/256 (0.00%)